#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "CudaKernel.h"

//jeah i should use some classes and stuff, but for first it was much simpler
//for me to just use some global variables
texture<float, 3,hipReadModeElementType>       CTtex;
hipArray                                       *d_volumeArray = 0;
float                                           CTmaxValue = 0;

texture<float, 3,hipReadModeElementType>       MRtex;
hipArray                                       *d_volumeArrayMR = 0;
float                                           MRmaxValue = 0;
float*                                          device_matrix_ptr = 0;
int                                             device_matrix_count = 0;

float*                                          device_result = NULL;
float*                                          device_output = NULL;
static float                                    sizefaktor = 2.0f;
static bool                                     volumeInitCT = false;
static bool                                     volumeInitMR = false;

void initCuda(){
    hipFree(0);
}

void generateCudaTexture(unsigned short* hostdata, int x, int y, int z, bool CT){
    if(volumeInitMR && !CT) return;
    if(volumeInitCT && CT) return;
    //if i want to use the full size...
    //i have to use a float buffer since CUDA does not support
    //the linear filter on int/short textures :(

    std::vector<float> dataasFloat;
    float max = 0;
    for(int i = 0; i < x*y*z;++i){
        dataasFloat.push_back((float)hostdata[i]);
        if(max < dataasFloat[i]){
            max = dataasFloat[i];
        }
    }

    std::vector<float> dataasFloatHalf;
    float valu = 0;
    float val = 0;
    for(int pz = 0; pz < z-1; pz+=2){
        for(int py = 0; py < y-1; py+=2){
            for(int px = 0; px < x-1; px+=2){
                //std::cout << px << " "<< py << " "<< pz << std::endl;
                val = hostdata[px + py*x + pz * x* y];
                val += hostdata[px + py*x + (pz+1)* x* y];
                val += hostdata[px + (py+1)*x + pz * x* y];
                val += hostdata[px + (py+1)*x + (pz+1) * x* y];
                val += hostdata[(px+1) + py*x + pz* x* y];
                val += hostdata[(px+1) + py*x + (pz+1)* x* y];
                val += hostdata[(px+1) + (py+1)*x + pz* x* y];
                val += hostdata[(px+1) + (py+1)*x + (pz+1)* x* y];
                val /= 8.0f;
                dataasFloatHalf.push_back(val);
                valu += val;
            }
        }
    }


    //creating the texture, only have the size since i resampled the data
    const hipExtent extend = make_hipExtent(x/2, y/2, z/2);

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

    if(CT){
        hipMalloc3DArray(&d_volumeArray, &channelDesc, extend);

        hipMemcpy3DParms copyParams = {0};
        copyParams.srcPtr   = make_hipPitchedPtr((void*)&(dataasFloatHalf[0]), extend.width*sizeof(float), extend.width, extend.height);
        copyParams.dstArray = d_volumeArray;
        copyParams.extent   = extend;
        copyParams.kind     = hipMemcpyHostToDevice;
        hipMemcpy3D(&copyParams);

        CTtex.normalized = false;
        CTtex.filterMode = hipFilterModePoint;
        CTtex.addressMode[0] = hipAddressModeClamp;
        CTtex.addressMode[1] = hipAddressModeClamp;
        CTtex.addressMode[2] = hipAddressModeClamp;
        CTtex.normalized = true;

        hipBindTextureToArray(CTtex, d_volumeArray, channelDesc);
        CTmaxValue = max;
        std::cout << "CT TEXTURE VALUE "<< (int)valu << std::endl;
        volumeInitCT = true;
    }else{
        hipMalloc3DArray(&d_volumeArrayMR, &channelDesc, extend);

        hipMemcpy3DParms copyParams = {0};
        copyParams.srcPtr   = make_hipPitchedPtr((void*)&(dataasFloatHalf[0]), extend.width*sizeof(float), extend.width, extend.height);
        copyParams.dstArray = d_volumeArrayMR;
        copyParams.extent   = extend;
        copyParams.kind     = hipMemcpyHostToDevice;
        hipMemcpy3D(&copyParams);

        MRtex.normalized = false;
        MRtex.filterMode = hipFilterModePoint;
        MRtex.addressMode[0] = hipAddressModeClamp;
        MRtex.addressMode[1] = hipAddressModeClamp;
        MRtex.addressMode[2] = hipAddressModeClamp;
        MRtex.normalized = true;

        hipBindTextureToArray(MRtex, d_volumeArrayMR, channelDesc);
        MRmaxValue = max;
        std::cout << "MR TEXTURE VALUE "<< (int)valu << std::endl;
        volumeInitMR = true;
    }

}

//DEVICE CODE --------------------------------------------------------
__global__ void substractCTandMR(float*     result,
                                 const float      maxX,
                                 const float      maxY,
                                 const float      maxZ,
                                 const float      CTMax,
                                 const float      MRMax,
                                 const float      m11,
                                 const float      m12,
                                 const float      m13,
                                 const float      m14,
                                 const float      m21,
                                 const float      m22,
                                 const float      m23,
                                 const float      m24,
                                 const float      m31,
                                 const float      m32,
                                 const float      m33,
                                 const float      m34){
    __shared__ float sdata[512];

    unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    unsigned int z = (blockIdx.z * blockDim.z) + threadIdx.z;


    float3 ctPosition;
    ctPosition.x = (float)(x)/maxX;
    ctPosition.y = (float)(y)/maxY;
    ctPosition.z = (float)(z)/maxZ;

    //read the value
    float valueCT = tex3D(CTtex,ctPosition.x,ctPosition.y,ctPosition.z);
    float valueMR = 0.0f;

    valueCT = valueCT/CTMax;
    if(valueCT > 0.5f )valueCT = 0;

    ctPosition.x -= 0.5f;
    ctPosition.y -= 0.5f;
    ctPosition.z -= 0.5f;

    unsigned long index = 0;
    float subresult = 0;
    float3 mrPosition;

    mrPosition.x = ctPosition.x*m11+ctPosition.y*m12+ctPosition.z*m13+1*m14+0.5f;
    mrPosition.y = ctPosition.x*m21+ctPosition.y*m22+ctPosition.z*m23+1*m24+0.5f;
    mrPosition.z = ctPosition.x*m31+ctPosition.y*m32+ctPosition.z*m33+1*m34+0.5f;

    if(mrPosition.x >= 0.0f && mrPosition.x <= 1.0f &&
       mrPosition.y >= 0.0f && mrPosition.y <= 1.0f &&
       mrPosition.z >= 0.0f && mrPosition.z <= 1.0f){

        //get value and normalize it
       valueMR = tex3D(MRtex,mrPosition.x,mrPosition.y,mrPosition.z);
       valueMR = valueMR/MRMax;
    }

    index = 0;
    index += z*(unsigned int)(maxX*maxY);
    index += y*(unsigned int)(maxX) + x;

    subresult = (valueMR-valueCT)*(valueMR-valueCT);

    //test use the thread reduction of the block to reduce the global memory acces
    int tid = threadIdx.z * blockDim.x *blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; // calculate the 1d index
    sdata[tid] = subresult;

    __syncthreads();
    for (unsigned int s=blockDim.x*blockDim.y*blockDim.z/2; s>0; s>>=1) {
    if (tid < s) {
    sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
    }
    if(tid == 0){
        index = blockIdx.z * 32 *32 + blockIdx.y * 32+ blockIdx.x;
        result[index] =sdata[0];
    }
    //result[index] = subresult;
}


__global__ void reduce(float* g_idata, float* g_odata){
    __shared__ float sdata[512];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
    __syncthreads();
    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
    if (tid < s) {
    sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
    }
    if(tid == 0){
        g_odata[blockIdx.x] = sdata[0];
    }
}


// HOST CODE -----------------------------------------------------------------------


static int count = 13;
static std::vector<float> result_vector;
void initDevice(int x, int y, int z){
    if(device_result == NULL)
        hipMalloc((void**) &device_result, sizeof(float)*x*y*z/512);

    if(device_output == NULL)
        hipMalloc((void**) &device_output, sizeof(float)*x*y*z/512);

    result_vector.resize(count);
}

float sumReduce(int x, int y, int z){
    int arraySize = x*y*z;
    float* temp = NULL;
    const int threads = 512;

    while(arraySize >= threads){
        arraySize = arraySize/threads/2;
        reduce<<<arraySize,threads>>>(device_result,device_output);
        temp = device_output;
        device_output = device_result;
        device_result = temp;
    }
    float* result = new float[arraySize];
    hipMemcpy(&(result[0]), temp, sizeof(float)*arraySize, hipMemcpyDeviceToHost);

    float resultVal = 0;
    for(int i = 0; i < arraySize;++i){
        resultVal += result[i];
    }
    return resultVal;
}


const std::vector<float>& step(int x, int y, int z,float* matrix){


    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks( x/threadsPerBlock.x,
                    y/threadsPerBlock.y,
                    z/threadsPerBlock.z);

    int matInd = 0;
    for(int i = 0; i < count;++i){
        matInd = i*16;
        substractCTandMR<<<numBlocks,threadsPerBlock>>>(device_result,
                                                        (float)x,
                                                        (float)y,
                                                        (float)z,
                                                        CTmaxValue,
                                                        MRmaxValue,
                                                        matrix[matInd],matrix[matInd+4],matrix[matInd+8],matrix[matInd+12],
                                                        matrix[matInd+1],matrix[matInd+5],matrix[matInd+9],matrix[matInd+13],
                                                        matrix[matInd+2],matrix[matInd+6],matrix[matInd+10],matrix[matInd+14]);
        hipDeviceSynchronize();
        result_vector[i] = sumReduce(x/8,y/8,z/8);
        hipDeviceSynchronize();
    }

    return result_vector;
}

