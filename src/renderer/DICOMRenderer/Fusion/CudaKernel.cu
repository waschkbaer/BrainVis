#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "CudaKernel.h"

//GLOBAL Varaibles , jeah there is no class here deal with it right now :(
texture<float, 3,hipReadModeElementType>       CTtex;
hipArray                                       *d_volumeArray = 0;
float                                           CTmaxValue = 0;

texture<float, 3,hipReadModeElementType>       MRtex;
hipArray                                       *d_volumeArrayMR = 0;
float                                           MRmaxValue = 0;
float*                                          device_matrix_ptr = 0;
int                                             device_matrix_count = 0;

float*                                          host_result = NULL;
float*                                          device_result = NULL;

static float                                    sizefaktor = 2.0f;

void initCuda(){
    hipFree(0);
}

void generateCudaTexture(unsigned short* hostdata, int x, int y, int z, bool CT){
    const hipExtent extend = make_hipExtent(x, y, z);

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

    std::vector<float> dataasFloat;
    float max = 0;
    for(int i = 0; i < x*y*z;++i){
        dataasFloat.push_back((float)hostdata[i]);
        if(max < dataasFloat[i]){
            max = dataasFloat[i];
        }
    }

    if(CT){
        for(int i = 0; i < x*y*z;++i){
            dataasFloat[i] = dataasFloat[i]/max;
            if(dataasFloat[i] > 0.5f){
                dataasFloat[i] = 0;
            }
        }


        hipMalloc3DArray(&d_volumeArray, &channelDesc, extend);

        hipMemcpy3DParms copyParams = {0};
        copyParams.srcPtr   = make_hipPitchedPtr((void*)&(dataasFloat[0]), extend.width*sizeof(float), extend.width, extend.height);
        copyParams.dstArray = d_volumeArray;
        copyParams.extent   = extend;
        copyParams.kind     = hipMemcpyHostToDevice;
        hipMemcpy3D(&copyParams);

        CTtex.normalized = false;
        CTtex.filterMode = hipFilterModePoint;
        CTtex.addressMode[0] = hipAddressModeClamp;
        CTtex.addressMode[1] = hipAddressModeClamp;
        CTtex.addressMode[2] = hipAddressModeClamp;
        CTtex.normalized = true;

        hipBindTextureToArray(CTtex, d_volumeArray, channelDesc);
        CTmaxValue = max;
    }else{
        for(int i = 0; i < x*y*z;++i){
            dataasFloat[i] = dataasFloat[i]/max;
        }

        hipMalloc3DArray(&d_volumeArrayMR, &channelDesc, extend);

        hipMemcpy3DParms copyParams = {0};
        copyParams.srcPtr   = make_hipPitchedPtr((void*)&(dataasFloat[0]), extend.width*sizeof(float), extend.width, extend.height);
        copyParams.dstArray = d_volumeArrayMR;
        copyParams.extent   = extend;
        copyParams.kind     = hipMemcpyHostToDevice;
        hipMemcpy3D(&copyParams);

        MRtex.normalized = false;
        MRtex.filterMode = hipFilterModePoint;
        MRtex.addressMode[0] = hipAddressModeClamp;
        MRtex.addressMode[1] = hipAddressModeClamp;
        MRtex.addressMode[2] = hipAddressModeClamp;
        MRtex.normalized = true;

        hipBindTextureToArray(MRtex, d_volumeArrayMR, channelDesc);
        MRmaxValue = max;
    }
}

inline __device__ float3 mul(float3* p, float* m){
    float3 v;
    v.x = p->x*m[0]+p->y*m[4]+p->z*m[8]+1*m[12]+0.5f;
    v.y = p->x*m[1]+p->y*m[5]+p->z*m[9]+1*m[13]+0.5f;
    v.z = p->x*m[2]+p->y*m[6]+p->z*m[10]+1*m[14]+0.5f;
    return v;
}


__global__ void subVolumes(float* result,
                           float* matrixptr,
                           float sizefaktor,
                           float xVolDim,
                           float yVolDim,
                           float zVolDim){
    float3 ctPos;
    ctPos.x = (float)blockIdx.x/(xVolDim-1.0f)*sizefaktor;
    ctPos.y = (float)threadIdx.x/(yVolDim-1.0f)*sizefaktor;
    ctPos.z = (float)blockIdx.z/(zVolDim-1.0f)*sizefaktor;

    float valueCT = tex3D(CTtex,ctPos.x,ctPos.y,ctPos.z);
    float valueMR = 0;

    ctPos.x -= 0.5f;
    ctPos.y -= 0.5f;
    ctPos.z -= 0.5f;

    float3 mrPos = mul(&ctPos, &matrixptr[blockIdx.y*16]);
    if(mrPos.x >= 0.0f && mrPos.x <= 1.0f &&
       mrPos.y >= 0.0f && mrPos.y <= 1.0f &&
       mrPos.z >= 0.0f && mrPos.z <= 1.0f){
       valueMR = tex3D(MRtex,mrPos.x,mrPos.y,mrPos.z);
    }

    float dif = (valueCT-valueMR);
    dif = dif*dif;
    int index = blockIdx.x + xVolDim*yVolDim*blockIdx.y + threadIdx.x*xVolDim;
    atomicAdd(&result[index],dif);
}

void setMatrixVector(float* matrix, int matrixcount){
    if(device_matrix_ptr == 0 || device_matrix_count != matrixcount){
        hipFree(device_matrix_ptr);
        hipMalloc((void**) &device_matrix_ptr, sizeof(float)*16*matrixcount);
    }
    hipMemcpy(device_matrix_ptr,matrix,sizeof(float)*16*matrixcount,hipMemcpyHostToDevice);
    device_matrix_count = matrixcount;
}

void setSizeFaktor(float sf){
    sizefaktor = sf;
}

const std::vector<float>&  subtractVolume(int x, int y, int z){
    dim3 grid(x/sizefaktor,device_matrix_count,z*sizefaktor);
    dim3 threadBlock(y/sizefaktor,1,1);

    if(host_result == NULL)
        host_result = new float[x*y*device_matrix_count];

    if(device_result == NULL)
        hipMalloc((void**) &device_result, sizeof(float)*x*y*device_matrix_count);

    memset(host_result,0,sizeof(float)*x*y*device_matrix_count);
    hipMemcpy(device_result,host_result,sizeof(float)*x*y*device_matrix_count, hipMemcpyHostToDevice);

    subVolumes<<<grid,threadBlock>>>(device_result,device_matrix_ptr,sizefaktor,(float)x,(float)y,(float)z);
    hipDeviceSynchronize();

    hipMemcpy(host_result, device_result, sizeof(float)*x*y*device_matrix_count, hipMemcpyDeviceToHost);

    std::vector<float> maxValues;
    maxValues.resize(device_matrix_count);
    int matsel = 0;
    for(int i = 0; i < x*y*device_matrix_count;++i){
        matsel = i/(x*y);
        maxValues[matsel] += host_result[i];
    }

    return maxValues;
}
