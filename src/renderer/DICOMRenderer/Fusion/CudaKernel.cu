#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "CudaKernel.h"


#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
        } \
    } while (0)

texture<float, 3,hipReadModeElementType>      CTtex;
hipArray                       *d_volumeArray = 0;
texture<float, 3,hipReadModeElementType>      MRtex;
hipArray                       *d_volumeArrayMR = 0;
float*                          device_matrix_ptr = 0;
int                             device_matrix_count = 0;
float*                          device_result = 0;
std::vector<float>              host_result;

void initCuda(){
    hipFree(0);
}

struct vec{
    float x;
    float y;
    float z;
    float w;
};

void generateCudaTexture(unsigned short* hostdata, int x, int y, int z, bool CT){
    const hipExtent extend = make_hipExtent(x, y, z);

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

    std::vector<float> dataasFloat;
    for(int i = 0; i < x*y*z;++i){
        dataasFloat.push_back((float)hostdata[i]);
    }

    if(CT){
        hipMalloc3DArray(&d_volumeArray, &channelDesc, extend);
        cudaCheckErrors("hipMalloc3D error");

        hipMemcpy3DParms copyParams = {0};
        copyParams.srcPtr   = make_hipPitchedPtr((void*)&(dataasFloat[0]), extend.width*sizeof(float), extend.width, extend.height);
        copyParams.dstArray = d_volumeArray;
        copyParams.extent   = extend;
        copyParams.kind     = hipMemcpyHostToDevice;
        hipMemcpy3D(&copyParams);
        cudaCheckErrors("hipMemcpy3D fail");

        CTtex.normalized = false;
        CTtex.filterMode = hipFilterModeLinear;
        CTtex.addressMode[0] = hipAddressModeClamp;
        CTtex.addressMode[1] = hipAddressModeClamp;
        CTtex.addressMode[2] = hipAddressModeClamp;
        CTtex.normalized = true;

        hipBindTextureToArray(CTtex, d_volumeArray, channelDesc);
        cudaCheckErrors("bind fail");
    }else{
        hipMalloc3DArray(&d_volumeArrayMR, &channelDesc, extend);
        cudaCheckErrors("hipMalloc3D error");

        hipMemcpy3DParms copyParams = {0};
        copyParams.srcPtr   = make_hipPitchedPtr((void*)&(dataasFloat[0]), extend.width*sizeof(float), extend.width, extend.height);
        copyParams.dstArray = d_volumeArrayMR;
        copyParams.extent   = extend;
        copyParams.kind     = hipMemcpyHostToDevice;
        hipMemcpy3D(&copyParams);
        cudaCheckErrors("hipMemcpy3D fail");

        MRtex.normalized = false;
        MRtex.filterMode = hipFilterModeLinear;
        MRtex.addressMode[0] = hipAddressModeClamp;
        MRtex.addressMode[1] = hipAddressModeClamp;
        MRtex.addressMode[2] = hipAddressModeClamp;
        MRtex.normalized = true;

        hipBindTextureToArray(MRtex, d_volumeArrayMR, channelDesc);
        cudaCheckErrors("bind fail");
    }
}

inline __device__ float3 mul(float3* p, float* m){
    float3 v;
    v.x = p->x*m[0]+p->y*m[4]+p->z*m[8]+1*m[12]+0.5f;
    v.y = p->x*m[1]+p->y*m[5]+p->z*m[9]+1*m[13]+0.5f;
    v.z = p->x*m[2]+p->y*m[6]+p->z*m[10]+1*m[14]+0.5f;
    return v;
}


__global__ void test(float* result, float* matrixptr){
    float3 ctPos;
    ctPos.x = (float)blockIdx.x/511.0f*2.0f;
    ctPos.y = (float)threadIdx.x/511.0f*2.0f;
    ctPos.z = (float)blockIdx.z/167.0f*2.0f;

    float valueCT = tex3D(CTtex,ctPos.x,ctPos.y,ctPos.z);
    valueCT= valueCT/4095.0f;
    if(valueCT >= 0.6f){
       valueCT = 0;
    }
    float valueMR = 0;

    ctPos.x -= 0.5f;
    ctPos.y -= 0.5f;
    ctPos.z -= 0.5f;

    float3 mrPos = mul(&ctPos, &matrixptr[blockIdx.y*16]);
    if(mrPos.x >= 0.0f && mrPos.x <= 1.0f &&
       mrPos.y >= 0.0f && mrPos.y <= 1.0f &&
       mrPos.z >= 0.0f && mrPos.z <= 1.0f){
       valueMR = tex3D(MRtex,mrPos.x,mrPos.y,mrPos.z);
       valueMR = valueMR/1019.0f;
    }

    float dif = ((valueCT-valueMR)*(valueCT-valueMR));
    int index = blockIdx.x + 512*blockIdx.y;
    atomicAdd(&result[index],dif);
}


void setMatrixVector(float* matrix, int matrixcount){
    if(device_matrix_ptr == 0 || device_matrix_count != matrixcount){
        hipFree(device_matrix_ptr);
        hipMalloc((void**) &device_matrix_ptr, sizeof(float)*16*matrixcount);
    }
    hipMemcpy(device_matrix_ptr,matrix,sizeof(float)*16*matrixcount,hipMemcpyHostToDevice);
    device_matrix_count = matrixcount;
}

void setEmptyResultVector(int size){
    if(device_result == 0 || size != host_result.size())
        hipMalloc((void**) &device_result, sizeof(float)*size);

    host_result.clear();
    host_result.resize(size);

    hipMemcpy(device_result, &host_result[0], sizeof(float)*size, hipMemcpyHostToDevice);
}

const std::vector<double>  subtractVolume(int x, int y, int z){
    dim3 grid(x/2,device_matrix_count,z/2);
    dim3 threadBlock(y/2,1,1);

    float* host_sumresult = new float[x*device_matrix_count];
    float* device_sumresult;
    memset(host_sumresult,0,sizeof(float)*x*device_matrix_count);
    hipMalloc((void**) &device_sumresult, sizeof(float)*x*device_matrix_count);
    hipMemcpy(device_sumresult,host_sumresult,sizeof(float)*x*device_matrix_count, hipMemcpyHostToDevice);

    test<<<grid,threadBlock>>>(device_sumresult,device_matrix_ptr);
    hipDeviceSynchronize();
    cudaCheckErrors("kernel fail");

    hipMemcpy(host_sumresult, device_sumresult, sizeof(float)*x*device_matrix_count, hipMemcpyDeviceToHost);

    host_result.clear();

    std::vector<double> maxValues;
    maxValues.resize(device_matrix_count);
    for(int i = 0; i < x*device_matrix_count;++i){
        maxValues[i/x] += host_sumresult[i];
    }

   /* for(double d : maxValues){
        std::cout << " val : "<< d << std::endl;
    }*/

    return maxValues;
}
