#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "CudaKernel.h"

//GLOBAL Varaibles , jeah there is no class here deal with it right now :(
texture<float, 3,hipReadModeElementType>       CTtex;
hipArray                                       *d_volumeArray = 0;
float                                           CTmaxValue = 0;

texture<float, 3,hipReadModeElementType>       MRtex;
hipArray                                       *d_volumeArrayMR = 0;
float                                           MRmaxValue = 0;
float*                                          device_matrix_ptr = 0;
int                                             device_matrix_count = 0;

float*                                          host_result = NULL;
float*                                          device_result = NULL;

static float                                    sizefaktor = 2.0f;

void initCuda(){
    hipFree(0);
}

void generateCudaTexture(unsigned short* hostdata, int x, int y, int z, bool CT){
    std::vector<float> dataasFloat;
    std::vector<float> dataasFloatHalf;
    std::vector<float> dataasFloatQuarter;
    float max = 0;
    for(int i = 0; i < x*y*z;++i){
        dataasFloat.push_back((float)hostdata[i]);
        if(max < dataasFloat[i]){
            max = dataasFloat[i];
        }
    }
    float val = 0;
    for(int pz = 0; pz < z-1; pz+=2){
        for(int py = 0; py < y-1; py+=2){
            for(int px = 0; px < x-1; px+=2){
                //std::cout << px << " "<< py << " "<< pz << std::endl;
                val = hostdata[px + py*x + pz * x* y];
                val += hostdata[px + py*x + (pz+1)* x* y];
                val += hostdata[px + (py+1)*x + pz * x* y];
                val += hostdata[px + (py+1)*x + (pz+1) * x* y];
                val += hostdata[(px+1) + py*x + pz* x* y];
                val += hostdata[(px+1) + py*x + (pz+1)* x* y];
                val += hostdata[(px+1) + (py+1)*x + pz* x* y];
                val += hostdata[(px+1) + (py+1)*x + (pz+1)* x* y];
                val /= 8.0f;
                dataasFloatHalf.push_back(val);
            }
        }
    }

    /*for(int pz = 0; pz < (z/2)-1; pz+=2){
        for(int py = 0; py < (y/2)-1; py+=2){
            for(int px = 0; px < (x/2)-1; px+=2){
                //std::cout << px << " "<< py << " "<< pz << std::endl;
                val = dataasFloatHalf[px + py*x/2 + pz * x/2* y/2];
                val += dataasFloatHalf[px + py*x/2 + (pz+1)* x/2* y/2];
                val += dataasFloatHalf[px + (py+1)*x/2 + pz * x/2* y/2];
                val += dataasFloatHalf[px + (py+1)*x/2 + (pz+1) * x/2* y/2];
                val += dataasFloatHalf[(px+1) + py*x/2 + pz* x/2* y/2];
                val += dataasFloatHalf[(px+1) + py*x/2 + (pz+1)* x/2* y/2];
                val += dataasFloatHalf[(px+1) + (py+1)*x/2 + pz* x/2* y/2];
                val += dataasFloatHalf[(px+1) + (py+1)*x/2 + (pz+1)* x/2* y/2];
                val /= 8.0f;
                dataasFloatQuarter.push_back(val);
            }
        }
    }

    std::cout << dataasFloat.size() << "   "<< dataasFloatHalf.size()<< " " << dataasFloatQuarter.size() <<std::endl;*/

    const hipExtent extend = make_hipExtent(x/2, y/2, z/2);

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

    if(CT){
        hipMalloc3DArray(&d_volumeArray, &channelDesc, extend);

        hipMemcpy3DParms copyParams = {0};
        copyParams.srcPtr   = make_hipPitchedPtr((void*)&(dataasFloatHalf[0]), extend.width*sizeof(float), extend.width, extend.height);
        copyParams.dstArray = d_volumeArray;
        copyParams.extent   = extend;
        copyParams.kind     = hipMemcpyHostToDevice;
        hipMemcpy3D(&copyParams);

        CTtex.normalized = false;
        CTtex.filterMode = hipFilterModePoint;
        CTtex.addressMode[0] = hipAddressModeClamp;
        CTtex.addressMode[1] = hipAddressModeClamp;
        CTtex.addressMode[2] = hipAddressModeClamp;
        CTtex.normalized = true;

        hipBindTextureToArray(CTtex, d_volumeArray, channelDesc);
        CTmaxValue = max;
    }else{
        hipMalloc3DArray(&d_volumeArrayMR, &channelDesc, extend);

        hipMemcpy3DParms copyParams = {0};
        copyParams.srcPtr   = make_hipPitchedPtr((void*)&(dataasFloatHalf[0]), extend.width*sizeof(float), extend.width, extend.height);
        copyParams.dstArray = d_volumeArrayMR;
        copyParams.extent   = extend;
        copyParams.kind     = hipMemcpyHostToDevice;
        hipMemcpy3D(&copyParams);

        MRtex.normalized = false;
        MRtex.filterMode = hipFilterModePoint;
        MRtex.addressMode[0] = hipAddressModeClamp;
        MRtex.addressMode[1] = hipAddressModeClamp;
        MRtex.addressMode[2] = hipAddressModeClamp;
        MRtex.normalized = true;

        hipBindTextureToArray(MRtex, d_volumeArrayMR, channelDesc);
        MRmaxValue = max;
    }
}

inline __device__ float3 mul(float3* p, float* m){
    float3 v;
    v.x = p->x*m[0]+p->y*m[4]+p->z*m[8]+1*m[12]+0.5f;
    v.y = p->x*m[1]+p->y*m[5]+p->z*m[9]+1*m[13]+0.5f;
    v.z = p->x*m[2]+p->y*m[6]+p->z*m[10]+1*m[14]+0.5f;
    return v;
}


__global__ void subVolumes(float* result,
                           float* matrixptr,
                           float sizefaktor,
                           float xVolDim,
                           float yVolDim,
                           float zVolDim,
                           float ctMax,
                           float mrMax){
    float3 ctPos;
    ctPos.x = (float)blockIdx.x/(xVolDim-1.0f)*sizefaktor;
    ctPos.y = (float)threadIdx.x/(yVolDim-1.0f)*sizefaktor;
    ctPos.z = (float)blockIdx.z/(zVolDim-1.0f)*sizefaktor;

    float valueCT = tex3D(CTtex,ctPos.x,ctPos.y,ctPos.z);
    valueCT = valueCT/ctMax;
    if(valueCT > 0.5f){
        valueCT = 0;
    }
    float valueMR = 0;

    ctPos.x -= 0.5f;
    ctPos.y -= 0.5f;
    ctPos.z -= 0.5f;

    float3 mrPos = mul(&ctPos, &matrixptr[blockIdx.y*16]);
    if(mrPos.x >= 0.0f && mrPos.x <= 1.0f &&
       mrPos.y >= 0.0f && mrPos.y <= 1.0f &&
       mrPos.z >= 0.0f && mrPos.z <= 1.0f){
       valueMR = tex3D(MRtex,mrPos.x,mrPos.y,mrPos.z);
       valueMR = valueMR/mrMax;
    }

    float dif = (valueCT-valueMR);
    dif = dif*dif;
    int index = blockIdx.x + xVolDim*yVolDim*blockIdx.y + threadIdx.x*xVolDim;
    atomicAdd(&result[index],dif);
}

void setMatrixVector(float* matrix, int matrixcount){
    if(device_matrix_ptr == 0 || device_matrix_count != matrixcount){
        hipFree(device_matrix_ptr);
        hipMalloc((void**) &device_matrix_ptr, sizeof(float)*16*matrixcount);
    }
    hipMemcpy(device_matrix_ptr,matrix,sizeof(float)*16*matrixcount,hipMemcpyHostToDevice);
    device_matrix_count = matrixcount;
}

void setSizeFaktor(float sf){
    sizefaktor = sf;
}

const std::vector<float>&  subtractVolume(int x, int y, int z){
    dim3 grid(x/sizefaktor,device_matrix_count,z/sizefaktor);
    dim3 threadBlock(y/sizefaktor,1,1);

    if(host_result == NULL)
        host_result = new float[x*y*device_matrix_count];

    if(device_result == NULL)
        hipMalloc((void**) &device_result, sizeof(float)*x*y*device_matrix_count);

    memset(host_result,0,sizeof(float)*x*y*device_matrix_count);
    hipMemcpy(device_result,host_result,sizeof(float)*x*y*device_matrix_count, hipMemcpyHostToDevice);

    subVolumes<<<grid,threadBlock>>>(device_result,device_matrix_ptr,sizefaktor,(float)x,(float)y,(float)z,CTmaxValue,MRmaxValue);
    hipDeviceSynchronize();

    hipMemcpy(host_result, device_result, sizeof(float)*x*y*device_matrix_count, hipMemcpyDeviceToHost);

    std::vector<float> maxValues;
    maxValues.resize(device_matrix_count);
    int matsel = 0;
    for(int i = 0; i < x*y*device_matrix_count;++i){
        matsel = i/(x*y);
        maxValues[matsel] += host_result[i];
    }

    return maxValues;
}
